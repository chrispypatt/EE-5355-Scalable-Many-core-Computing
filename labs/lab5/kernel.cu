
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512
#define WARP_SIZE 32
#define NUM_WARPS (BLOCK_SIZE/WARP_SIZE)

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY 128

/******************************************************************************
 GPU kernels
*******************************************************************************/

__global__ void gpu_global_queuing_kernel(unsigned int *nodePtrs,
  unsigned int *nodeNeighbors, unsigned int *nodeVisited,
  unsigned int *currLevelNodes, unsigned int *nextLevelNodes,
  unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //large number of nodes is too many for our threads so each thread may have to do more than one
  for (idx=idx; idx < *numCurrLevelNodes; idx += gridDim.x * blockDim.x){
    unsigned int node = currLevelNodes[idx];
    for(unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      //check if node was visited, if it wasn't, flag it as visited and update queue
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      unsigned int visited = atomicExch(&(nodeVisited[neighbor]), 1); 
      if(!visited){
        //increment numNextLevelNodes and use old value as index for this node's place in the fringe
        unsigned int gq_idx = atomicAdd(numNextLevelNodes,1);
        nextLevelNodes[gq_idx] = neighbor;
      }
    }
  }
}

__global__ void gpu_block_queuing_kernel(unsigned int *nodePtrs,
  unsigned int *nodeNeighbors, unsigned int *nodeVisited,
  unsigned int *currLevelNodes, unsigned int *nextLevelNodes,
  unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {
  
  //setup block's shared queue
  __shared__ unsigned int s_nextLevelNodes[BQ_CAPACITY], s_numNextLevelNodes, s_start;

  if (threadIdx.x == 0) s_numNextLevelNodes = 0; //init block's numNExtLevelNodes
  __syncthreads();

  // INSERT KERNEL CODE HERE
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  //large number of nodes is too many for our threads so each thread may have to do more than one
  for (idx=idx; idx < *numCurrLevelNodes; idx += gridDim.x * blockDim.x){
    unsigned int node = currLevelNodes[idx];
    for(unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      //check if node was visited, if it wasn't, flag it as visited and update queue
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      unsigned int visited = atomicExch(&(nodeVisited[neighbor]), 1); 
      if(!visited){
        //increment numNextLevelNodes and use old value as index for this node's place in the fringe
        unsigned int bq_idx = atomicAdd(&s_numNextLevelNodes,1);
        if (bq_idx < BQ_CAPACITY){//make sure there is room in block queue
          s_nextLevelNodes[bq_idx] = neighbor;
        }else{//if not, put right into global queue
          s_numNextLevelNodes = BQ_CAPACITY;//s_numNextLevelNodes >= BQ_CAPACITY so reset to BQ_CAPACITY
          unsigned int gq_idx = atomicAdd(numNextLevelNodes,1);
          nextLevelNodes[gq_idx] = neighbor;
        }
      }
    }
  }
  __syncthreads();//wait for entire block to finish

  //update global numNextLevelNodes for other blocks to determine their start 
  if (threadIdx.x == 0){
    s_start = atomicAdd(numNextLevelNodes, s_numNextLevelNodes);
  }
  __syncthreads();

  for (unsigned int i = threadIdx.x; i < s_numNextLevelNodes; i += blockDim.x){
    nextLevelNodes[i+s_start] = s_nextLevelNodes[i];
  }
}



__global__ void gpu_warp_queuing_kernel(unsigned int *nodePtrs,
  unsigned int *nodeNeighbors, unsigned int *nodeVisited,
  unsigned int *currLevelNodes, unsigned int *nextLevelNodes,
  unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  //setup block's shared queue
  __shared__ unsigned int b_nextLevelNodes[BQ_CAPACITY], b_numNextLevelNodes, b_start;

  //setup warp queues
  __shared__ unsigned int w_nextLevelNodes[WQ_CAPACITY][NUM_WARPS];//allows for coalescing later
  __shared__ unsigned int w_numNextLevelNodes[NUM_WARPS], w_start[NUM_WARPS];
  unsigned int warp_idx = threadIdx.x % NUM_WARPS;

  //init block's numNextLevelNodes
  if (threadIdx.x == 0) b_numNextLevelNodes = 0; 
  //init each warp queue's numNextLevelNodes
  if (threadIdx.x < NUM_WARPS) w_numNextLevelNodes[threadIdx.x] = 0;  
  __syncthreads();

  //large number of nodes is too many for our threads so each thread may have to do more than one
  for (idx=idx; idx < *numCurrLevelNodes; idx += gridDim.x * blockDim.x){
    unsigned int node = currLevelNodes[idx];
    for(unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      //check if node was visited, if it wasn't, flag it as visited and update queue
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      unsigned int visited = atomicExch(&(nodeVisited[neighbor]), 1); 
      if(!visited){
        //increment numNextLevelNodes and use old value as index for this node's place in the fringe
        unsigned int wq_idx = atomicAdd(&(w_numNextLevelNodes[warp_idx]),1);
        if(wq_idx < WQ_CAPACITY){//make sure there is room in this thread's warp queue
          w_nextLevelNodes[wq_idx][warp_idx] = neighbor;
        }else{//if not, fall back to block and global queues
          w_numNextLevelNodes[warp_idx] = WQ_CAPACITY;//w_numNextLevelNodes >= WQ_CAPACITY so reset to WQ_CAPACITY
          unsigned int bq_idx = atomicAdd(&b_numNextLevelNodes,1);
          if (bq_idx < BQ_CAPACITY){//make sure there is room in block queue
            b_nextLevelNodes[bq_idx] = neighbor;
          }else{//if not, put right into global queue
            b_numNextLevelNodes = BQ_CAPACITY;//s_numNextLevelNodes >= BQ_CAPACITY so reset to BQ_CAPACITY
            unsigned int gq_idx = atomicAdd(numNextLevelNodes,1);
            nextLevelNodes[gq_idx] = neighbor;
          }
        }
      }
    }
  }
  __syncthreads();//wait for entire block to finish

  //update block's numNextLevelNodes so other warps can determine their start
  unsigned int offset = threadIdx.x/NUM_WARPS;
  if (offset == 0){//only first thread in a warp 
    w_start[warp_idx] = atomicAdd(&b_numNextLevelNodes, w_numNextLevelNodes[warp_idx]);
  }
  __syncthreads();
  //let each thread in the warp move elements from warp queue to block queue in coalesced fashion
  for (unsigned int i = offset; i < w_numNextLevelNodes[warp_idx]; i += WARP_SIZE){
    unsigned int bq_idx = w_start[warp_idx] + i;//w_numNextLevelNodes[warp_idx];
    if (bq_idx < BQ_CAPACITY){//make sure there is room in block queue
      b_nextLevelNodes[bq_idx] = w_nextLevelNodes[i][warp_idx];
    }else{//if not, put right into global queue
      b_numNextLevelNodes = BQ_CAPACITY;//s_numNextLevelNodes >= BQ_CAPACITY so reset to BQ_CAPACITY
      unsigned int gq_idx = atomicAdd(numNextLevelNodes,1);
      nextLevelNodes[gq_idx] = w_nextLevelNodes[i][warp_idx];
    }
  }
  __syncthreads();


  //update global numNextLevelNodes for other blocks to determine their start 
  if (threadIdx.x == 0){
    b_start = atomicAdd(numNextLevelNodes, b_numNextLevelNodes);
  }
  __syncthreads();

  for (unsigned int i = threadIdx.x; i < b_numNextLevelNodes; i += blockDim.x){
    nextLevelNodes[i+b_start] = b_nextLevelNodes[i];
  }
}

/******************************************************************************
 Functions
*******************************************************************************/

void cpu_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
  unsigned int *nodeVisited, unsigned int *currLevelNodes,
  unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
  unsigned int *numNextLevelNodes) {

  // Loop over all nodes in the curent level
  for(unsigned int idx = 0; idx < *numCurrLevelNodes; ++idx) {
    unsigned int node = currLevelNodes[idx];
    // Loop over all neighbors of the node
    for(unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1];
      ++nbrIdx) {
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      // If the neighbor hasn't been visited yet
      if(!nodeVisited[neighbor]) {
        // Mark it and add it to the queue
        nodeVisited[neighbor] = 1;
        nextLevelNodes[*numNextLevelNodes] = neighbor;
        ++(*numNextLevelNodes);
      }
    }
  }

}

void gpu_global_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
  unsigned int *nodeVisited, unsigned int *currLevelNodes,
  unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
  unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queuing_kernel <<< numBlocks , BLOCK_SIZE >>> (nodePtrs,
    nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
    numCurrLevelNodes, numNextLevelNodes);

}

void gpu_block_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
  unsigned int *nodeVisited, unsigned int *currLevelNodes,
  unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
  unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queuing_kernel <<< numBlocks , BLOCK_SIZE >>> (nodePtrs,
    nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
    numCurrLevelNodes, numNextLevelNodes);

}

void gpu_warp_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
  unsigned int *nodeVisited, unsigned int *currLevelNodes,
  unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
  unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queuing_kernel <<< numBlocks , BLOCK_SIZE >>> (nodePtrs,
    nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
    numCurrLevelNodes, numNextLevelNodes);

}

