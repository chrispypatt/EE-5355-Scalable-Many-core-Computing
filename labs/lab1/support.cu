#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdlib.h>
#include <stdio.h>

#include "support.h"

void initVector(unsigned int **vec_h, unsigned int size, unsigned int max)
{
    *vec_h = (unsigned int*)malloc(size*sizeof(unsigned int));

    if(*vec_h == NULL) {
        FATAL("Unable to allocate host");
    }

    for (unsigned int i=0; i < size; i++) {
        (*vec_h)[i] = (rand()%(max + 1));
    }

}

void verify(unsigned int* in, unsigned int* out, unsigned int num_in, unsigned int num_out) {

  // Initialize reference
  unsigned int* out_ref = (unsigned int*) malloc(num_out*sizeof(unsigned int));
  for(unsigned int outIdx = 0; outIdx < num_out; ++outIdx) {
      out_ref[outIdx] = 0;
  }

  // Compute reference out
  for(unsigned int inIdx = 0; inIdx < num_in; ++inIdx) {
      unsigned int intermediate = outInvariant(in[inIdx]);
      for(unsigned int outIdx = 0; outIdx < num_out; ++outIdx) {
          out_ref[outIdx] += outDependent(intermediate, inIdx, outIdx);
      }
  }

  // Compare to reference out
  for(unsigned int outIdx = 0; outIdx < num_out; ++outIdx) {
      if(out[outIdx] != out_ref[outIdx]) {
        printf("TEST FAILED at output index %u, reference = %u, computed = %u"
          "\n\n", outIdx, out_ref[outIdx], out[outIdx]);
        exit(0);
      }
  }
  printf("TEST PASSED\n\n");

  free(out_ref);

}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

__host__ __device__ unsigned int outInvariant(unsigned int inValue) {
    return inValue*inValue;
}

__host__ __device__ unsigned int outDependent(unsigned int value, unsigned int inIdx,
        unsigned int outIdx) {
    if(inIdx == outIdx) {
        return 2*value;
    } else if(inIdx > outIdx) {
        return value/(inIdx - outIdx);
    } else {
        return value/(outIdx - inIdx);
    }
}

// Allocate a device array of same size as data.
unsigned int* allocateDeviceArray(unsigned int* data, int num_elements){
	int size = num_elements * sizeof(unsigned int);
	unsigned int* d_data = data;
	hipError_t cuda_ret = hipMalloc((void**) &d_data, size);
	if(cuda_ret != hipSuccess) {
		printf("Unable to allocate device memory");
		exit(0);
	}
	return d_data;
}

// Copy a host array to a device array.
void copyToDeviceArray(unsigned int* d_data, const unsigned int*  h_data, int num_elements)
{
    int size = num_elements * sizeof(unsigned int);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
}

// Copy a device array to a host array.
void copyFromDeviceArray(unsigned int* h_data, const unsigned int*  d_data, int num_elements)
{
    int size = num_elements * sizeof(unsigned int);
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

}

