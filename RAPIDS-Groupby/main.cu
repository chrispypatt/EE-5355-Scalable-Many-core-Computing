//
//  main.cpp
//  RAPIDS
//
//  Created by Aaron on 11/19/18.
//  Copyright © 2018 Aaron Nightingale. All rights reserved.
//
//  This is a simple CPU groupby class (currently only MAX is implemented)
//  one key column and one value column.

#include <iostream>
#include <algorithm>
#include <chrono>
#include <vector>
#include <string>
#include "cpuGroupby.h"
#include "groupby.cu"

int main(int argc, const char * argv[]) {
        

	using Time = std::chrono::high_resolution_clock;
	using fsec = std::chrono::duration<float>;

        int num_rows = 100000;
        int num_key_cols = 2;
        int num_val_cols = 3;
        int num_distinct_keys = 10;
	std::vector<std::string> args(argv, argv+argc);
        if (argc == 2){
	  num_rows = stoi(args.at(1));
        } else if(argc == 4){
	  num_rows = stoi(args.at(1));
	  num_key_cols = stoi(args.at(2));
	  num_val_cols = stoi(args.at(3));
        } else if(argc == 5){
	  num_rows = stoi(args.at(1));
	  num_key_cols = stoi(args.at(2));
	  num_val_cols = stoi(args.at(3));
	  num_distinct_keys = stoi(args.at(4));
        } else {
	  if (argc != 1) {
	    std::cerr << "Invalid arguments" << std::endl;
	    exit(1);
	  }
	}
        // Setting up the CPU groupby
        cpuGroupby slowGroupby(num_key_cols, num_val_cols, num_rows);

        slowGroupby.fillRand(num_distinct_keys, num_rows);

        int *original_key_columns;
	hipHostMalloc((void**)&original_key_columns, sizeof(int)*num_key_cols*num_rows);
        int *original_value_columns;
	hipHostMalloc((void**)&original_value_columns, sizeof(int)*num_val_cols*num_rows);
        std::copy(slowGroupby.key_columns, slowGroupby.key_columns + num_key_cols*num_rows, original_key_columns);
        std::copy(slowGroupby.value_columns, slowGroupby.value_columns + num_val_cols*num_rows, original_value_columns);
        
        auto start = Time::now();

        slowGroupby.groupby();

        auto end = Time::now(); 
        fsec cpu_duration = end - start;

        // Insert GPU function calls here...
        int *gpu_output_keys, *gpu_output_values;
        int gpu_output_rows = 0;
        gpu_output_keys = new int[slowGroupby.num_key_rows*slowGroupby.num_key_columns];
        gpu_output_values = new int[slowGroupby.num_value_rows*slowGroupby.num_value_columns];

        start = Time::now();

        groupby_GPU(original_key_columns, slowGroupby.num_key_columns,
                slowGroupby.num_key_rows, original_value_columns, 
                slowGroupby.num_value_columns, slowGroupby.num_value_rows, 
                slowGroupby.ops, slowGroupby.num_ops,
                gpu_output_keys, gpu_output_values, gpu_output_rows);
        end = Time::now();
        
        slowGroupby.printGPUResults(gpu_output_keys, gpu_output_values);

        fsec gpu_duration = end - start;

	std::cout << "CPU time: " << cpu_duration.count() << " s" << std::endl;
	std::cout << "GPU time: " << gpu_duration.count() << " s" << std::endl;

        slowGroupby.validGPUResult(gpu_output_keys, gpu_output_values, gpu_output_rows);

        hipHostFree(original_value_columns);
        hipHostFree(original_key_columns);
        return 0;
}
